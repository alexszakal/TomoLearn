#include "hip/hip_runtime.h"
#include <Gen1CT.hpp>

#include <config.h>
#include <cstdio>
#include <chrono>

#if ENABLE_CUDA

#include <hip/hip_runtime.h>
//#include <>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

/***
 * Ray driven projection (Hao Gao method) KERNEL
 * @param phantom Pointer to the phantom data
 * @param sinogram Pointer to the resulting sinogram
 * @return Returns void
 */
__global__
void rayDrivenProjectionKernel(const double* phantom, int numberOfPixelsX, int numberOfPixelsY, double pixSizesX, double pixSizesY,
		             double* sinogram, int numAngles, double* angles, int numDetPixels, double detPixSize ){

	int detPixIdx = blockIdx.x*blockDim.x + threadIdx.x;
	int angIdx =    blockIdx.y*blockDim.y + threadIdx.y;

	//printf("\n detPixIdx: %d, angIdx: %d", detPixIdx, angIdx);

	if ( detPixIdx >= numDetPixels){
		return;
	}
	if (angIdx >= numAngles){
		return;
	}

	int sinogramDataIdx = angIdx*numDetPixels + detPixIdx; //Column-major order!

	double halfPhantomWidth = numberOfPixelsX*pixSizesX/2;
	double halfPhantomHeight = numberOfPixelsY*pixSizesY/2;

	//trigonometric functions of the angles
	double theta = angles [angIdx];
	double sinTheta, cosTheta;
	sincos(theta, &sinTheta, &cosTheta);

	//Distance of the detector plane from origin which is outside of the phantom
	double detDist = 1.1 * sqrt(pow(pixSizesX*numberOfPixelsX, 2) + pow(pixSizesY*numberOfPixelsY, 2) ); //Distance between the detector plane and centre of rotation

	double p1[2];
	double p2[2];

	double sinoPointValue=0.0;  //Temporary local variable to accumulate result

	//beam intersects the columns at most two pixels
	if( pixSizesY / pixSizesX >= std::abs(std::tan(M_PI/2-theta)) ){
		const double invPixSize1 = 1 / pixSizesY;
		const double pixSizeRatio01 = pixSizesX / pixSizesY;

	    const double t = -1*numDetPixels*detPixSize/2+(detPixIdx+0.5)*detPixSize;

	    p1[0]=detDist * sinTheta + t * cosTheta;
	    p1[1]=-1*detDist * cosTheta + t * sinTheta;

	    p2[0] = -1 * detDist * sinTheta + t * cosTheta;
	    p2[1] = detDist * cosTheta + t * sinTheta;

    	const double ky = (p1[1]-p2[1])/(p1[0]-p2[0]);
    	const double pathInSinglePixel = sqrt(1+ky*ky)*pixSizesX;

    	double yi_minus = ( halfPhantomHeight - (ky*( (-1)   *pixSizesX - halfPhantomWidth - p1[0] ) + p1[1] ) ) * invPixSize1;
    	const double yi_minusIncrement = -1*ky*invPixSize1*pixSizesX;
    	const double yi_plusIncrement = -1 * ky*pixSizeRatio01;
    	//go through the columns of the image
    	for(int colIdx=0; colIdx<numberOfPixelsX; ++colIdx){
    		yi_minus += yi_minusIncrement;
    		double yi_plus = yi_minus + yi_plusIncrement;

    		int Yi_minusIdx = floor(yi_minus);
	    	int Yi_plusIdx = floor(yi_plus);

	    	double l_minus, l_plus; //intersecting lengths when crossing two pixels
	    	if( Yi_minusIdx == Yi_plusIdx ){ //intersecting only one pixel
	    		if( (Yi_minusIdx < numberOfPixelsY) and (Yi_minusIdx >= 0 ) ){
	    			sinoPointValue += pathInSinglePixel * phantom[Yi_minusIdx*numberOfPixelsX + colIdx];
	    		}
	    	}
	    	else{
	    		if ( (Yi_minusIdx < numberOfPixelsY) and (Yi_minusIdx >= 0) ){
	    			l_minus=(max(Yi_minusIdx, Yi_plusIdx)-yi_minus) / (yi_plus - yi_minus) * pathInSinglePixel;

	    			sinoPointValue += l_minus * phantom[Yi_minusIdx*numberOfPixelsX + colIdx];
	    			//We have l_minus -> we can calculate l_plus with only a subtraction from pathInSinglePixel
	    			if( (Yi_plusIdx < numberOfPixelsY) and (Yi_plusIdx >= 0 ) ){
	    				sinoPointValue += (pathInSinglePixel- l_minus) * phantom[Yi_plusIdx*numberOfPixelsX + colIdx];
	    				continue;
	    			}
	    		}

	    		if ( (Yi_plusIdx < numberOfPixelsY) and (Yi_plusIdx >= 0 ) ){
	    			l_plus=(yi_plus - max(Yi_minusIdx, Yi_plusIdx)) / (yi_plus - yi_minus) * pathInSinglePixel;

	    			sinoPointValue += l_plus * phantom[Yi_plusIdx*numberOfPixelsX + colIdx];
	    		}
	    	}
	    }
	}
	else{      //beam intersects the rows at most two pixels
		const double invPixSize0 = 1 / pixSizesX;
		const double pixSizeRatio10 = pixSizesY / pixSizesX;

		const double t = -1*numDetPixels*detPixSize/2+(detPixIdx+0.5)*detPixSize;

	    p1[0]=detDist * sinTheta + t * cosTheta;
	    p1[1]=-1*detDist * cosTheta + t * sinTheta;

	    p2[0] = -1 * detDist * sinTheta + t * cosTheta;
	    p2[1] = detDist * cosTheta + t * sinTheta;

	   	const double kx = (p1[0]-p2[0])/(p1[1]-p2[1]);
	   	const double pathInSinglePixel = sqrt(1+kx*kx)*pixSizesY;

	   	//go through the rows of the image
	   	double xi_minus = (halfPhantomWidth + (kx*( halfPhantomHeight - (-1)*pixSizesY - p1[1] ) + p1[0] ) )  * invPixSize0;
	   	const double xi_minusIncrement = -1*kx*invPixSize0*pixSizesY;
	   	const double xi_plusIncrement = -1*kx*pixSizeRatio10;
	    for(int rowIdx=0; rowIdx<numberOfPixelsY; ++rowIdx){
	    	xi_minus += xi_minusIncrement;
	    	double xi_plus = xi_minus + xi_plusIncrement;

	    	int Xi_minusIdx = floor(xi_minus);
	        int Xi_plusIdx = floor(xi_plus);

	        double l_minus, l_plus; //intersecting lengths
	        if( Xi_minusIdx == Xi_plusIdx ){
	        	if( (Xi_minusIdx < numberOfPixelsX) and (Xi_minusIdx >= 0 ) ){
	        		sinoPointValue += pathInSinglePixel * phantom[rowIdx*numberOfPixelsX + Xi_minusIdx];
	        	}
	        }
	        else{
	        	if ( (Xi_minusIdx < numberOfPixelsX) and (Xi_minusIdx >= 0 ) ){
	        		l_minus=(max(Xi_minusIdx, Xi_plusIdx)-xi_minus) / (xi_plus - xi_minus) * pathInSinglePixel;

	        		sinoPointValue += l_minus * phantom[rowIdx*numberOfPixelsX + Xi_minusIdx];
	        		//We have l_minus -> we can calculate l_plus with only a subtraction from pathInSinglePixel
	        		if ( (Xi_plusIdx <= numberOfPixelsX) and (Xi_plusIdx >= 0 ) ){   //Shortcut to avoid one more std::max call
	        			sinoPointValue += (pathInSinglePixel-l_minus) * phantom[rowIdx*numberOfPixelsX + Xi_minusIdx];
	        			continue;
	        		}
	        	}

	    		if ( (Xi_plusIdx < numberOfPixelsX) and (Xi_plusIdx >= 0) ){
	    			l_plus=(xi_plus - max(Xi_minusIdx, Xi_plusIdx)) / (xi_plus - xi_minus) * pathInSinglePixel;

	    			sinoPointValue += l_plus * phantom[rowIdx*numberOfPixelsX + Xi_plusIdx];
	        	}
	        }
	    }
	}

	sinogram[sinogramDataIdx] = sinoPointValue; //Write the result to global memory

}

void Gen1CT::printGpuParameters(){
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
	    hipDeviceProp_t prop;
	    hipGetDeviceProperties(&prop, i);
	    printf("Device Number: %d\n", i);
	    printf("  Device name: %s\n", prop.name);
	    printf("  Memory Clock Rate (KHz): %d\n",
	           prop.memoryClockRate);
	    printf("  Memory Bus Width (bits): %d\n",
	           prop.memoryBusWidth);
	    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
	           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}
}

void launchRayDrivenProjectionKernel(const double* phantomData, std::array<int, 2> numberOfPixels, std::array<double, 2> pixSizes,
		                   int numAngles, const double* anglesData, int pixNum, double detWidth,
						   double* sinogramData){

	auto start = std::chrono::high_resolution_clock::now();

	//Allocate space and copy data
	double *d_phantom;
	checkCudaErrors(hipMalloc(&d_phantom, sizeof(double) * numberOfPixels[0]*numberOfPixels[1] ));
	checkCudaErrors(hipMemcpy(d_phantom, phantomData,
			                   sizeof(double) * numberOfPixels[0]*numberOfPixels[1], hipMemcpyHostToDevice));

	double *d_sinogram;
	checkCudaErrors(hipMalloc(&d_sinogram, sizeof(double) * numAngles * pixNum ));

	double *d_angles;
	checkCudaErrors(hipMalloc(&d_angles, sizeof(double) * numAngles ));
	checkCudaErrors(hipMemcpy(d_angles, anglesData,
					                   sizeof(double) * numAngles, hipMemcpyHostToDevice));

	auto phase1 = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds >(phase1 - start);
	std::cout << "\nMemory allocation and data transfer to GPU took " << duration.count() << " milliseconds";

	//CALL THE PROJECTION KERNEL!!!!
	const dim3 blockSize(16,16);
	const dim3 gridSize(pixNum/blockSize.x+1,numAngles/blockSize.y+1);
	rayDrivenProjectionKernel<<<gridSize, blockSize>>>(d_phantom, numberOfPixels[0], numberOfPixels[1], pixSizes[0], pixSizes[1],
				                                 d_sinogram, numAngles, d_angles, pixNum, detWidth/pixNum);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	auto phase2 = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::milliseconds >(phase2 - phase1);
	std::cout << "\n Running the rayDriven projection kernel on GPU took " << duration.count() << " milliseconds";

	//Read back the result and free memory
	checkCudaErrors(hipMemcpy(sinogramData, d_sinogram,
					                   sizeof(double) * numAngles*pixNum, hipMemcpyDeviceToHost));

	hipFree(d_sinogram);
	hipFree(d_phantom);
	hipFree(d_angles);

	auto phase3 = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::milliseconds >(phase3 - phase2);
	std::cout << "\nCopy back results and free GPU memory took " << duration.count() << " milliseconds";

}

__global__
void rayDrivenBackprojectionKernel(double* d_sinogram, int numAngles, double* d_angles, double* sinThetaVector, double* cosThetaVector, int pixNum, double detWidth,
									double* d_backProjection, int numberOfPixelsX, int numberOfPixelsY, double resolutionX, double resolutionY){

	int xIdx = blockIdx.x*blockDim.x + threadIdx.x;
	int yIdx = blockIdx.y*blockDim.y + threadIdx.y;

	if ( xIdx >= numberOfPixelsX){
		return;
	}
	if ( yIdx >= numberOfPixelsY){
		return;
	}

	int backProjectionDataIdx = yIdx*numberOfPixelsX + xIdx; //Column-major order!

	const double pixelRadius = sqrt(pow(resolutionX/2,2) + pow(resolutionY/2,2)); //Radius of circle drawn around a pixel
    const double halfDetWidth = detWidth / 2;
    const double invDetPixSize= pixNum/detWidth;

    double xValue= -1*numberOfPixelsX*resolutionX/2 + resolutionX/2 + xIdx*resolutionX;
    double yValue=    numberOfPixelsY*resolutionY/2 - resolutionY/2 - yIdx*resolutionY;

    const double pixelSize = detWidth/pixNum;

    d_backProjection[backProjectionDataIdx] = 0.0;

	//Calculate the value in pixel
	//Go through the angles
	for(int angIdx=0; angIdx<numAngles; ++angIdx){
		//Determine the contributing detector pixels (i.e. rays)
		double xr = xValue*cosThetaVector[static_cast<size_t>(angIdx)]+
				       yValue*sinThetaVector[static_cast<size_t>(angIdx)];   //This corresponds to "t" of the image pixel center

		double minPixIdx = (xr + halfDetWidth - pixelRadius )*invDetPixSize;
		double maxPixIdx = (xr + halfDetWidth + pixelRadius )*invDetPixSize;

		//Calculate the intersection length and add to the backprojected image
		//Go through the possible pixels
		double lSum=0;
		double angleContrib = 0;
		double a = cosThetaVector[static_cast<size_t>(angIdx)];
		double b = sinThetaVector[static_cast<size_t>(angIdx)];
		double absa = abs(a);
		double absb = abs(b);

		//Calculate the boundaries of pixel intersection types
		double d_max = (std::abs(a*resolutionX) + std::abs(b*resolutionY))/2; //For definition see HaoGao's article
		double d_min =  std::abs( std::abs(a*resolutionX) - std::abs(b*resolutionY) )/2;

		for(int detPixIdx=max(0,static_cast<int>(minPixIdx));
			    detPixIdx <= min(static_cast<int>(maxPixIdx)+0, static_cast<int>(pixNum-1) );
			    ++detPixIdx){
			double c = -1*halfDetWidth+(detPixIdx+0.5)*pixelSize;  // "t" of the detector pixel center

			//Calculate the intersection length
			double d_act=std::abs(a*xValue + b*yValue - c);
			double l;

			if(d_act < d_min){
				if( absa < absb ){
					l=resolutionX/absb;
				}
				else{
					l=resolutionY/absa;
				}
			}
			else if( (d_act >= d_min) and (d_act < d_max) ){
				l=(d_max-d_act)/absa/absb;
			}
			else{
				l=0;
			}

			lSum += l;
			angleContrib += d_sinogram[angIdx*pixNum+detPixIdx]*l;
		}
		if(lSum != 0){
			d_backProjection[backProjectionDataIdx] += angleContrib/lSum;
		}
	}

	//Multiply with dTheta
	d_backProjection[backProjectionDataIdx] = d_backProjection[backProjectionDataIdx]*M_PI/numAngles;
}

void launchRayDrivenBackprojectionKernel(const double* sinogram, int numAngles, const double* anglesData, int pixNum, double detWidth,
										 double* backProjection, const std::array<int,2>& numberOfPixels, const std::array<double,2>& resolution){

	auto start = std::chrono::high_resolution_clock::now();

	double *d_sinogram;
	checkCudaErrors(hipMalloc(&d_sinogram, sizeof(double) * numAngles*pixNum ));
	checkCudaErrors(hipMemcpy(d_sinogram, sinogram,
				                   sizeof(double) * numAngles*pixNum, hipMemcpyHostToDevice));

	double *d_angles;
	checkCudaErrors(hipMalloc(&d_angles, sizeof(double) * numAngles ));
	checkCudaErrors(hipMemcpy(d_angles, anglesData,
					                   sizeof(double) * numAngles, hipMemcpyHostToDevice));

	double *d_backprojection;
	checkCudaErrors(hipMalloc(&d_backprojection, sizeof(double) * numberOfPixels[0]*numberOfPixels[1] ));

	std::vector<double>	thetaVector,
	                    sinThetaVector,
						cosThetaVector;

	//Vector of theta values and trigonometric functions
	for(int i=0; i<numAngles; i++){
	  	thetaVector.push_back( std::fmod(anglesData[i], 2*M_PI) );
	   	sinThetaVector.push_back( sin(thetaVector[static_cast<size_t>(i)]) );
	   	cosThetaVector.push_back( cos(thetaVector[static_cast<size_t>(i)]) );
	}

	double *d_sinThetaVector;
	checkCudaErrors(hipMalloc(&d_sinThetaVector, sizeof(double) * numAngles ));
	checkCudaErrors(hipMemcpy(d_sinThetaVector, sinThetaVector.data(),
						                   sizeof(double) * numAngles, hipMemcpyHostToDevice));

	double *d_cosThetaVector;
	checkCudaErrors(hipMalloc(&d_cosThetaVector, sizeof(double) * numAngles ));
	checkCudaErrors(hipMemcpy(d_cosThetaVector, cosThetaVector.data(),
						                   sizeof(double) * numAngles, hipMemcpyHostToDevice));

	auto stop1 = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop1 - start);
	std::cout << "Phase1 took " << duration.count() << " milliseconds" << std::endl;

	auto start2 = std::chrono::high_resolution_clock::now();
	const dim3 blockSize(16,16);
	const dim3 gridSize(numberOfPixels[0]/blockSize.x+1,numberOfPixels[1]/blockSize.y+1);
	rayDrivenBackprojectionKernel<<<gridSize, blockSize>>>(d_sinogram, numAngles, d_angles, d_sinThetaVector, d_cosThetaVector, pixNum, detWidth,
			d_backprojection, numberOfPixels[0], numberOfPixels[1], resolution[0], resolution[1]);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	auto stop2 = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop2 - start2);
	std::cout << "Running the kernel took " << duration.count() << " milliseconds" << std::endl;

	auto start3 = std::chrono::high_resolution_clock::now();
	//Read back the result and free memory
	checkCudaErrors(hipMemcpy(backProjection, d_backprojection,
						         sizeof(double) * numberOfPixels[0]*numberOfPixels[1], hipMemcpyDeviceToHost));

	hipFree(d_sinogram);
	hipFree(d_angles);
	hipFree(d_backprojection);

	auto stop3 = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop3 - start3);
	std::cout << "Reading the results and free memory took " << duration.count() << " milliseconds" << std::endl;

}
#endif




